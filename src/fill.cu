
#include <hip/hip_runtime.h>
#include <numeric>

constexpr int blocksize = 100;

__global__ void fillStuff(int *in) {
	in[threadIdx.x] = threadIdx.x;
}

__global__ void runStuff(int *in, int *out) {
	out[threadIdx.x] = in[threadIdx.x] * 2;
}

long runFill() {
	dim3 block{blocksize, 1};
	dim3 grid{1, 1};

	int *in, *out;

	hipMalloc((void**)&in, blocksize * sizeof(int));
	hipMalloc((void**)&out, blocksize * sizeof(int));

	fillStuff<<<grid, block>>>(in);
	runStuff<<<grid, block>>>(in, out);

	int result[blocksize];

	hipMemcpy(result, out, blocksize, hipMemcpyHostToDevice);

	hipFree(in);
	hipFree(out);
	
	return std::accumulate(result, result + blocksize, static_cast<long>(0));
}
